#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_fp16.h>
#include "vec_add.h"

// CUDA kernel for vector addition using half-precision
__global__ void vectorAdd(const half* A, const half* B, half* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = __hadd(A[i], B[i]);
    }
}

// Helper function to check CUDA errors
void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

int main() {
    int N = 1000;
    size_t size = N * sizeof(half);

    // Allocate host memory
    half *h_A = new half[N];
    half *h_B = new half[N];
    half *h_C = new half[N];

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = __float2half(static_cast<float>(i));
        h_B[i] = __float2half(static_cast<float>(i) * 2);
    }

    // Allocate device memory
    half *d_A, *d_B, *d_C;
    checkCudaErrors(hipMalloc((void**)&d_A, size));
    checkCudaErrors(hipMalloc((void**)&d_B, size));
    checkCudaErrors(hipMalloc((void**)&d_C, size));

    // Copy vectors from host memory to device memory
    checkCudaErrors(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    // Launch the vector addition kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Check for kernel launch errors
    checkCudaErrors(hipGetLastError());

    // Copy result vector from device memory to host memory
    checkCudaErrors(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    // Verify the result
    for (int i = 0; i < N; i++) {
        float expected = static_cast<float>(i) + static_cast<float>(i) * 2;
        float result = __half2float(h_C[i]);
        if (abs(result - expected) > 0.001) {
            std::cerr << "Result verification failed at element " << i << "!" << std::endl;
            exit(-1);
        }
    }

    std::cout << "Test PASSED" << std::endl;

    // Free device memory
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}