#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector

#include "src/kernels/cal_padding_offset.h"

// this kernel is only int type input and output, not fp32 or half
// we compare the kernel correctnesss by eyes and result print infos
// `./paddingoffset` to run
int main() {
    const int batch_size = 4;
    const int max_q_len = 5;
    // debug info, better to retain: std::cout <<"batch_size=" << batch_size << "  vocab_size=" << vocab_size << std::endl;
    int *h_seq_lens, *d_seq_lens;
    h_seq_lens = (int *)malloc(sizeof(int) * batch_size);
    hipMalloc((void **) &d_seq_lens, sizeof(int) * batch_size);

    int *h_cum_seqlens, *d_cum_seqlens;
    h_cum_seqlens = (int *)malloc(sizeof(int) * (batch_size + 1));
    hipMalloc((void **) &d_cum_seqlens, sizeof(int) * (batch_size + 1));
    
    int *h_padding_offset, *d_padding_offset;
    h_padding_offset = (int *)malloc(sizeof(int) * batch_size * max_q_len);
    hipMalloc((void **) &d_padding_offset, sizeof(int) * batch_size * max_q_len);

    for(int i = 0; i < batch_size; i++) {
       h_seq_lens[i] = batch_size - (i * i % 3);
    }
    hipMemcpy(d_seq_lens, h_seq_lens, sizeof(int) * batch_size, hipMemcpyHostToDevice);
    DataType type_int = getTensorType<int>();
    auto padding_offset = new TensorWrapper<int> {Device::GPU, type_int, {batch_size, max_q_len}, d_padding_offset};
    auto cum_seqlens = new TensorWrapper<int> {Device::GPU, type_int, {batch_size + 1}, d_cum_seqlens};
    auto input_lengths = new TensorWrapper<int> {Device::GPU, type_int, {batch_size}, d_seq_lens};
    
    // printf("before launch cuda kernel!\n");
    launchCalPaddingOffset(padding_offset, cum_seqlens, input_lengths);
    // hipDeviceSynchronize();
    // printf("after launch cuda kernel!\n");
    
    hipMemcpy(h_padding_offset, d_padding_offset, sizeof(int) * batch_size * max_q_len, hipMemcpyDeviceToHost);
    hipMemcpy(h_cum_seqlens, d_cum_seqlens, sizeof(int) * (batch_size + 1), hipMemcpyDeviceToHost);

    for (int i = 0; i < batch_size; ++i) {
        printf("%d ", h_seq_lens[i]);
    }
    printf("\n");

    printf("padding offset:\n");
    for (int i = 0; i < batch_size * max_q_len; ++i) {
        printf("%d ", h_padding_offset[i]);
    }
    printf("\n");

    printf("cum_seqlens:\n");
    for (int i = 0; i < batch_size + 1; ++i) {
        printf("%d ", h_cum_seqlens[i]);
    }
    printf("\n");
    //expected result is:
    // padding_offset: 0,0,0,2,2,2,4,4,4,0.... shape = [batchsize, max_q_len]
    // cum_seqlens: 0,3,6,9. shape=[batchsize+1]
    // debug info, better to retain: std::cout << "before free" << std::endl;
    free(h_seq_lens);
    free(h_padding_offset);
    free(h_cum_seqlens);
    hipFree(d_seq_lens);
    hipFree(d_padding_offset);
    hipFree(d_cum_seqlens);
    return 0;
}