#include "hip/hip_runtime.h"
#include <algorithm>      // std::fill_n
#include <cmath>         // expf, log
#include <cstdlib>       // rand
#include <iostream>      // snprintf
#include <string>        // std::string
#include <vector>        // std::vector

#include "src/kernels/includes/fused_decoder_self_attention.h"
#include "src/utils/macro.h"

// Bug1: MUST add CHECK to hipMemcpy to see if it works well
// Note: This CPU implementation still has bugs.
// When implementing LLM inference on CPU, reuse the CPU kernel to test its correctness.
// Use `./test_fused_decoder_attention` to test fp32 kernel
template <typename T>
void CPUMaskedAttention(
    T *q,
    T *k,
    T *v,
    T *k_cache,
    T *v_cache,
    float *mha_output,
    const int batch_size,
    const int num_heads,
    const int head_size,
    const int step
) {
    const int batch_stride = num_heads * head_size;
    const int head_stride = head_size;
    const int cache_offset = batch_size * batch_stride;
    const int block_nums = batch_size * num_heads;
    const float scale = rsqrt(float(head_size));

    const T *q_mem = q;
    const T *k_mem = k;
    const T *v_mem = v;

    // Temp buffer
    float *shared_qk = static_cast<float *>(malloc(sizeof(float) * (block_nums * (3 * head_size + step))));
    float *sq = shared_qk;
    float *sk = sq + block_nums * head_size;
    float *logits = sk + block_nums * head_size;
    float *sv = logits + block_nums * step;

    for (int batch_id = 0; batch_id < batch_size; ++batch_id) {
        for (int head_id = 0; head_id < num_heads; ++head_id) {
            float row_max = 0.0f;

            for (int iter = 0; iter < step; ++iter) {
                float attn_score = 0.0f;

                for (int tid = 0; tid < head_size; ++tid) {
                    const int qkv_offset = batch_id * batch_stride + head_id * head_stride + tid;

                    // Note: sq and sk's offset should be qkv_offset, not tid
                    sk[qkv_offset] = static_cast<float>(k_cache[iter * cache_offset + qkv_offset]);

                    // When final step, update k cache
                    if (iter == step - 1) {
                        // TODO: update k cache with k with bias add
                        k_cache[iter * cache_offset + qkv_offset] = k_mem[qkv_offset];
                        sk[qkv_offset] = static_cast<float>(k_mem[qkv_offset]);
                    }

                    sq[qkv_offset] = static_cast<float>(q_mem[qkv_offset]);
                    const float qk = sq[qkv_offset] * sk[qkv_offset] * scale;
                    attn_score += qk;
                }

                logits[batch_id * num_heads * step + head_id * step + iter] = attn_score;
                row_max = std::max(attn_score, row_max);
            }

            float fenzi = 0.0f;
            float fenmu = 0.0f;

            for (int iter = 0; iter < step; ++iter) {
                fenzi = expf(logits[batch_id * num_heads * step + head_id * step + iter] - row_max);
                fenmu += fenzi;
            }

            for (int iter = 0; iter < step; ++iter) {
                logits[batch_id * num_heads * step + head_id * step + iter] = fenzi / fenmu;
            }

            for (int tid = 0; tid < head_size; ++tid) {
                float O = 0.0f;
                const int qkv_offset = batch_id * batch_stride + head_id * head_stride + tid;

                for (int iter = 0; iter < step; ++iter) {
                    sv[qkv_offset] = static_cast<float>(v_cache[iter * cache_offset + qkv_offset]);

                    if (iter == step - 1) {
                        v_cache[iter * cache_offset + qkv_offset] = v_mem[qkv_offset];
                        sv[qkv_offset] = static_cast<float>(v_mem[qkv_offset]);
                    }

                    O += sv[qkv_offset] * logits[batch_id * num_heads * step + head_id * step + iter];
                }

                mha_output[qkv_offset] = O;
            }
        }
    }

    free(shared_qk);
}

template <typename T>
bool checkResult(
    const float *CPUoutput,
    const T *GPUoutput,
    const int output_size
) {
    for (int i = 0; i < output_size; ++i) {
        const float GPUres = static_cast<float>(GPUoutput[i]);
        if (fabs(CPUoutput[i] - GPUres) > 1e-6) {
            printf("The %dth result is wrong: CPUoutput = %f, GPUoutput = %f\n", i, CPUoutput[i], GPUres);
            return false;
        }
    }
    return true;
}

int main(int argc, char *argv[]) {
    constexpr int batch_size = 1;
    constexpr int head_size = 4;
    constexpr int num_heads = 2;
    constexpr int kv_num_heads = 2;
    constexpr int max_seq_len = 4;
    const int h_step = 4;
    const int h_layer_id = 0;
    const int rotary_embedding_dim = 128;
    const float rotary_embedding_base = 10000;
    const int max_position_embeddings = 2048;
    const bool use_dynamic_ntk = false; // for dyn scaling rope

    float *h_qkv, *d_qkv;
    const int qkv_size = batch_size * (2 * kv_num_heads + num_heads) * head_size;
    h_qkv = static_cast<float *>(malloc(sizeof(float) * qkv_size));
    hipMalloc(reinterpret_cast<void **>(&d_qkv), sizeof(float) * qkv_size);

    float *h_kcache, *d_kcache;
    const int kcache_size = max_seq_len * batch_size * kv_num_heads * head_size;
    h_kcache = static_cast<float *>(malloc(sizeof(float) * kcache_size));
    hipMalloc(reinterpret_cast<void **>(&d_kcache), sizeof(float) * kcache_size);

    float *h_vcache, *d_vcache;
    const int vcache_size = max_seq_len * batch_size * kv_num_heads * head_size;
    h_vcache = static_cast<float *>(malloc(sizeof(float) * vcache_size));
    hipMalloc(reinterpret_cast<void **>(&d_vcache), sizeof(float) * vcache_size);

    for (int i = 0; i < qkv_size; ++i) {
        if (i < batch_size * num_heads * head_size) {
            if (i < batch_size * num_heads * head_size / 2) {
                h_qkv[i] = static_cast<float>(i + 1);
            } else {
                h_qkv[i] = static_cast<float>(i - 3) / 10;
            }
        } else if (i < batch_size * (num_heads + kv_num_heads) * head_size) {
            if (i < batch_size * (num_heads + kv_num_heads / 2) * head_size) {
                h_qkv[i] = static_cast<float>(i + 5);
            } else {
                h_qkv[i] = static_cast<float>(i + 1) / 10;
            }
        } else if (i < batch_size * (num_heads + kv_num_heads * 2) * head_size) {
            if (i < batch_size * (num_heads + kv_num_heads + kv_num_heads / 2) * head_size) {
                h_qkv[i] = static_cast<float>(i - 3);
            } else {
                h_qkv[i] = static_cast<float>(i - 7) / 10;
            }
        }
        printf("h_qkv[%d] = %f\n", i, h_qkv[i]);
    }

    float *h_q = h_qkv;
    float *h_k = h_q + batch_size * num_heads * head_size;
    float *h_v = h_k + batch_size * (kv_num_heads + num_heads) * head_size;

    for (int i = 0; i < (kcache_size * h_step) / max_seq_len; ++i) {
        if (i < kcache_size / 2) {
            h_kcache[i] = static_cast<float>(i + 1);
            h_vcache[i] = static_cast<float>(i + 1);
        } else {
            h_kcache[i] = static_cast<float>(i - kcache_size / 2 + 1) / 10;
            h_vcache[i] = static_cast<float>(i - kcache_size / 2 + 1) / 10;
        }
        printf("h_kcache[%d] = %f\n", i, h_kcache[i]);
        printf("h_vcache[%d] = %f\n", i, h_vcache[i]);
    }

    float *h_o, *d_o;
    const int o_size = batch_size * num_heads * head_size;
    h_o = static_cast<float *>(malloc(sizeof(float) * o_size));
    hipMalloc(reinterpret_cast<void **>(&d_o), sizeof(float) * o_size);

    bool *h_finished = static_cast<bool *>(malloc(sizeof(bool) * batch_size));
    bool *d_finished;
    hipMalloc(reinterpret_cast<void **>(&d_finished), sizeof(bool) * batch_size);

    for (int i = 0; i < batch_size; ++i) {
        h_finished[i] = false;
    }

    float *h_qkv_bias = static_cast<float *>(malloc(sizeof(float) * (2 * kv_num_heads + num_heads) * head_size));
    float *d_qkv_bias;
    hipMalloc(reinterpret_cast<void **>(&d_qkv_bias), sizeof(float) * (2 * kv_num_heads + num_heads) * head_size);

    for (int i = 0; i < (2 * kv_num_heads + num_heads) * head_size; ++i) {
        h_qkv_bias[i] = 0.0f;
    }

    hipMemcpy(d_qkv, h_qkv, sizeof(float) * qkv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_qkv_bias, h_qkv_bias, sizeof(float) * (2 * kv_num_heads + num_heads) * head_size, hipMemcpyHostToDevice);
    hipMemcpy(d_finished, h_finished, sizeof(bool) * batch_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kcache, h_kcache, sizeof(float) * kcache_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vcache, h_vcache, sizeof(float) * vcache_size, hipMemcpyHostToDevice);

    DataType type = getTensorType<float>();
    DataType type_bool = getTensorType<bool>();
    DataType type_int = getTensorType<int>();

    TensorWrapper<float> *qkv = new TensorWrapper<float>(
        Device::GPU, type, 
        {batch_size, num_heads + 2 * kv_num_heads, head_size}, 
        d_qkv
    );
    TensorWrapper<float> *kcache = new TensorWrapper<float>(
        Device::GPU, type, 
        {h_layer_id, batch_size, kv_num_heads, max_seq_len, head_size}, 
        d_kcache
    );
    TensorWrapper<float> *vcache = new TensorWrapper<float>(
        Device::GPU, type, 
        {h_layer_id, batch_size, kv_num_heads, max_seq_len, head_size}, 
        d_vcache
    );
    TensorWrapper<bool> *finished = new TensorWrapper<bool>(Device::GPU, type_bool, {batch_size}, d_finished);
    TensorWrapper<int> *step = new TensorWrapper<int>(Device::CPU, type_int, {1}, const_cast<int *>(&h_step));
    TensorWrapper<int> *layer_id = new TensorWrapper<int>(Device::CPU, type_int, {1}, const_cast<int *>(&h_layer_id));
    TensorWrapper<float> *mha_output = new TensorWrapper<float>(Device::GPU, type, {batch_size, num_heads, head_size}, d_o);

    BaseWeight<float> qkv_weight;
    qkv_weight.bias = d_qkv_bias;

    LlamaAttentionStaticParams params;
    params.rotary_embedding_dim = rotary_embedding_dim;
    params.rotary_embedding_base = rotary_embedding_base;
    params.max_position_embeddings = max_position_embeddings;
    params.use_dynamic_ntk = use_dynamic_ntk;

    launchDecoderMaskedMultiHeadAttention(
        qkv,
        &qkv_weight,
        layer_id,
        kcache,
        vcache,
        finished,
        step,
        mha_output,
        &params
    );

    CHECK(hipMemcpy(h_o, d_o, sizeof(float) * o_size, hipMemcpyDeviceToHost));

    float *CPU_output = static_cast<float *>(malloc(sizeof(float) * o_size));
    CPUMaskedAttention<float>(h_q, h_k, h_v, h_kcache, h_vcache, CPU_output, batch_size, num_heads, head_size, h_step);

    const bool is_true = checkResult<float>(CPU_output, h_o, o_size);
    if (is_true) {
        printf("Test passed\n");
    } else {
        printf("Test failed\n");
    }

    free(h_qkv);
    free(h_kcache);
    free(h_vcache);
    free(h_o);
    free(CPU_output);
    free(h_finished);
    hipFree(d_finished);
    hipFree(d_qkv);
    hipFree(d_o);
    hipFree(d_kcache);
    hipFree(d_vcache);

    return 0;
}
