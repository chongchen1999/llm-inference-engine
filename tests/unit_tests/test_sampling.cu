#include <iostream>
#include "src/kernels/includes/sampling.h"
#include "src/utils/macro.h"

// Note: There is no CPU implementation of this kernel
// We compare the kernel correctness by eye and result print info
// Use `./test_sampling` to test fp32 GPU kernel
// Use `./test_sampling 1` to test fp16 GPU kernel

template <typename dtype>
void launch_sampling(int batch_size, int K, int step, int vocab_size, int end_id) {
    int *h_topkid = (int *)malloc(sizeof(int) * batch_size * K);
    int *d_topkid;
    hipMalloc((void **)&d_topkid, sizeof(int) * batch_size * K);

    dtype *h_topkval = (dtype *)malloc(sizeof(dtype) * batch_size * K);
    dtype *d_topkval;
    hipMalloc((void **)&d_topkval, sizeof(dtype) * batch_size * K);

    int *h_outid = (int *)malloc(sizeof(int) * batch_size);
    int *d_outid;
    hipMalloc((void **)&d_outid, sizeof(int) * batch_size);

    int *h_cuseqlen = (int *)malloc(sizeof(int) * batch_size);
    int *d_cuseqlen;
    hipMalloc((void **)&d_cuseqlen, sizeof(int) * batch_size);

    bool *h_finished = (bool *)malloc(sizeof(bool) * batch_size);
    bool *d_finished;
    hipMalloc((void **)&d_finished, sizeof(bool) * batch_size);

    for (int i = 0; i < batch_size; ++i) {
        h_finished[i] = false;
        h_cuseqlen[i] = 4;
    }

    for (int i = 0; i < batch_size * K; ++i) {
        h_topkid[i] = i;
        h_topkval[i] = (dtype)(K - 1 - (i % K));
    }

    CHECK(hipMemcpy(d_topkval, h_topkval, sizeof(dtype) * batch_size * K, hipMemcpyHostToDevice));
    DataType type = getTensorType<dtype>();
    TensorWrapper<dtype> *topk_val = new TensorWrapper<dtype>(Device::GPU, type, {batch_size, K}, d_topkval);

    CHECK(hipMemcpy(d_topkid, h_topkid, sizeof(int) * batch_size * K, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_cuseqlen, h_cuseqlen, sizeof(int) * batch_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_finished, h_finished, sizeof(bool) * batch_size, hipMemcpyHostToDevice));

    DataType type_int = getTensorType<int>();
    DataType type_bool = getTensorType<bool>();

    TensorWrapper<int> *topk_id = new TensorWrapper<int>(Device::GPU, type_int, {batch_size, K}, d_topkid);
    TensorWrapper<int> *cuseqlen = new TensorWrapper<int>(Device::GPU, type_int, {batch_size}, d_cuseqlen);
    TensorWrapper<bool> *finished = new TensorWrapper<bool>(Device::GPU, type_bool, {batch_size}, d_finished);
    TensorWrapper<int> *output_id = new TensorWrapper<int>(Device::GPU, type_int, {batch_size}, d_outid);

    IntDict intParams;
    intParams.insert({"step", step});
    intParams.insert({"vocab_size", vocab_size});
    intParams.insert({"end_id", end_id});

    std::cout << "Before launching sampling kernel" << std::endl;
    launchSampling<dtype>(topk_id, topk_val, cuseqlen, finished, output_id, intParams);
    std::cout << "After launching sampling kernel" << std::endl;

    std::cout << "Copying data from device to host" << std::endl;
    CHECK(hipMemcpy(h_outid, output_id->data, sizeof(int) * batch_size, hipMemcpyDeviceToHost));

    for (int i = 0; i < batch_size; ++i) {
        std::cout << "Sequence " << i + 1 << ": " << h_outid[i] << std::endl;
    }

    free(h_topkid);
    free(h_topkval);
    free(h_finished);
    free(h_cuseqlen);
    free(h_outid);
    hipFree(d_topkid);
    hipFree(d_topkval);
    hipFree(d_finished);
    hipFree(d_cuseqlen);
    hipFree(d_outid);
}

int main(int argc, char *argv[]) {
    const int batch_size = 3;
    const int K = 3;
    int vocab_size = 1000;
    int step = 6;
    int end_id = 10;

    if (argc > 1) {
        launch_sampling<half>(batch_size, K, step, vocab_size, end_id);
    } else {
        launch_sampling<float>(batch_size, K, step, vocab_size, end_id);
    }

    return 0;
}