#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>
#include "../utils/cuda_debug_utils.cuh"
#include "includes/decoder_self_attention.h"

// kv cache shape = [numlayers, bs, kv head num, max_seq_len, head size]
// bug1: scale's dtype must be float, not int
// bug2: mha_kernel_params struct's pointer is on CPU, not GPU, which causes we don't run the CUDA kernel, so add cudacheck is a must!
// bug3: blockreduce res should use tid=0 to write into smem
// bug4: GQA, kv_head_num brd to head_num, we can automatically do this by head id index like lmdeploy
// half or float version: the logits and mha output both are fp32 type, q k v are all accessed vectorizedly

template <typename T>
struct SumOp {
    __device__ __forceinline__ T operator()(const T &a, const T &b) const {
        return a + b;
    }
};

template <typename T>
struct MaxOp {
    __device__ __forceinline__ T operator()(const T &a, const T &b) const {
        return a > b ? a : b;
    }
};

template <template <typename> class ReductionOp, typename T>
__device__ __forceinline__ T warpReduce(T val) {
    #pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}

template <template <typename> class ReductionOp, typename T>
__device__ T blockReduce(T val) {
    const int tid = threadIdx.x;
    const int warp_id = tid >> 5;
    const int lane_id = tid & 31;
    const int warp_nums = (blockDim.x + 31) >> 5;
    static __shared__ T warp[32]; // threads in a block must be less than 1024
    val = warpReduce<ReductionOp, T>(val);
    if (lane_id == 0) {
        warp[warp_id] = val;
    }
    __syncthreads();
    const T warp_val = tid < warp_nums ? warp[tid] : 0;
    return warpReduce<ReductionOp, T>(warp_val);
}

/*
    dim3 grid(head_num * batch_size);
    dim3 block(head_size);
*/
template <typename T>
__global__ void maskedMultiHeadAttention(
    T *const q, // [bs, head_num, 1, head_size]
    T *const k, // [bs, kv_head_num, 1, head_size]
    T *const v, // [bs, kv_head_num, 1, head_size]
    const T *const qkv_bias, // bias, [qkv_head_num * head_size]
    T *const k_cache, // [layer_num, batch_size, kv_num_heads, max_seq_len, head_size]
    T *const v_cache, // [layer_num, batch_size, kv_num_heads, max_seq_len, head_size]
    T *const mha_output, // [batch_size, num_heads, head_size]
    const int batch_size, 
    const int head_num, 
    const int kv_head_num,
    const int max_seq_len, 
    const int head_size, 
    const int step,
    const int rotary_embedding_dim, 
    const float rotary_embedding_base
) {
    const int q_batch_id = blockIdx.x / head_num;
    const int q_head_id = blockIdx.x % head_num;
    const int tid = threadIdx.x;

    const int kv_batch_id = q_batch_id;
    const int repeated_kv_heads = head_num / kv_head_num;
    const int kv_head_id = q_head_id / repeated_kv_heads;

    const int batch_stride = head_num * head_size;
    const int kv_batch_stride = kv_head_num * head_size;
    const int head_stride = head_size;
    const int q_offset = q_batch_id * batch_stride + q_head_id * head_stride + tid;
    const int k_offset = kv_batch_id * kv_batch_stride + kv_head_id * head_stride + tid;

    const int vec_size = Vec<T>::size;
    const int vec_q_offset = q_batch_id * batch_stride + q_head_id * head_stride + tid * vec_size;
    const int vec_kv_offset = kv_batch_id * kv_batch_stride + kv_head_id * head_stride + tid * vec_size;
    const int cache_offset = kv_batch_id * kv_head_num * max_seq_len * head_size +
                             kv_head_id * max_seq_len * head_size + tid * vec_size;

    const int step_stride = head_size;
    const float scale = rsqrt(static_cast<float>(head_size));
    using Vec_t = typename Vec<T>::Type;

    extern __shared__ char shared_qk[];
    T *const shared_q = reinterpret_cast<T *>(shared_qk);
    float *const logits = reinterpret_cast<float *>(shared_q + head_size);
    Vec_t *const vec_shared_q = reinterpret_cast<Vec_t *>(shared_q);

    Vec_t &vec_q = *reinterpret_cast<Vec_t *>(q + vec_q_offset);
    Vec_t &vec_k = *reinterpret_cast<Vec_t *>(k + vec_kv_offset);
    Vec_t &vec_v = *reinterpret_cast<Vec_t *>(v + vec_kv_offset);

    if (tid * vec_size < head_size) {
        if (qkv_bias != nullptr) {
            const Vec_t q_bias = *reinterpret_cast<const Vec_t *>(qkv_bias + q_head_id * head_size + tid * vec_size);
            const Vec_t k_bias = *reinterpret_cast<const Vec_t *>(qkv_bias + (head_num + kv_head_id) * head_size + tid * vec_size);
            const Vec_t v_bias = *reinterpret_cast<const Vec_t *>(qkv_bias + (head_num + kv_head_num + kv_head_id) * head_size + tid * vec_size);
            VectorizedOperator<Vec_t>::add_assign(vec_q, q_bias);
            VectorizedOperator<Vec_t>::add_assign(vec_k, k_bias);
            VectorizedOperator<Vec_t>::add_assign(vec_v, v_bias);
        }
        vec_shared_q[tid] = vec_q;
    }
    __syncthreads();

    const Vec_t vec_zero = ScalarCast2Vector::scalar_cast2_vector<Vec_t, float>(0.0f);
    const Vec_t vec_scale = ScalarCast2Vector::scalar_cast2_vector<Vec_t, float>(scale);

    *reinterpret_cast<Vec_t *>(k_cache + (step - 1) * step_stride + cache_offset) = vec_k;
    #pragma unroll
    for (int kv_id = 0; kv_id < step; ++kv_id) {
        Vec_t vec_cached_k = vec_zero;
        Vec_t vec_qkT = vec_zero; // q * K^T
        if (tid * vec_size < head_size) {
            vec_cached_k = *reinterpret_cast<Vec_t *>(k_cache + kv_id * step_stride + cache_offset);
            vec_qkT = VectorizedOperator<Vec_t>::mul(vec_shared_q[tid], vec_cached_k);
            VectorizedOperator<Vec_t>::mul_assign(vec_qkT, vec_scale);
        }
        
        T qk_acc = vec_qkT.x + vec_qkT.y + vec_qkT.z + vec_qkT.w;
        T attention_score = blockReduce<SumOp, T>(qk_acc);
        if (tid == 0) {
            logits[kv_id] = attention_score;
        }
        __syncthreads();
    }

    const T local_logit = tid < step ? static_cast<T>(logits[tid]) : 0;
    __shared__ float row_max, sum_exp;
    const T block_max = blockReduce<MaxOp, T>(local_logit);
    if (tid == 0) {
        row_max = block_max;
    }
    __syncthreads();

    const T cur_exp = tid < step ? expf(local_logit - row_max) : 0;
    const T block_sum_exp = blockReduce<SumOp, T>(cur_exp);
    if (tid == 0) {
        sum_exp = block_sum_exp + 1e-6f;
    }
    __syncthreads();

    if (tid < step) {
        logits[tid] = static_cast<T>(cur_exp / sum_exp);
    }
    __syncthreads();

    if (tid * vec_size < head_size) {
        Vec_t vec_attention_score = ScalarCast2Vector::scalar_cast2_vector<Vec_t, T>(0.0f);
        *reinterpret_cast<Vec_t *>(v_cache + (step - 1) * step_stride + cache_offset) = vec_v;

        #pragma unroll
        for (int kv_id = 0; kv_id < step; ++kv_id) {
            Vec_t vec_cached_v = *reinterpret_cast<Vec_t *>(v_cache + kv_id * step_stride + cache_offset);
            VectorizedOperator<Vec_t>::add_assign(
                vec_attention_score, 
                VectorizedOperator<Vec_t>::mul(
                    vec_cached_v, 
                    ScalarCast2Vector::scalar_cast2_vector<Vec_t, float>(logits[kv_id])
                )
            );
        }
        *reinterpret_cast<Vec_t *>(mha_output + q_offset) = vec_attention_score;
    }
}

template <>
__global__ void maskedMultiHeadAttention(
    half *const q, 
    half *const k, 
    half *const v, 
    const half *const qkv_bias, 
    half *const k_cache, 
    half *const v_cache, 
    half *const mha_output,
    const int batch_size, 
    const int head_num, 
    const int kv_head_num,
    const int max_seq_len, 
    const int head_size, 
    const int step,
    const int rotary_embedding_dim, 
    const float rotary_embedding_base
) {
    // Note: To sync with newest fp32 MHA
}

template <typename T>
void launchDecoderMaskedMultiHeadAttention(
    TensorWrapper<T> *qkv_buf, // [bs, qkv_head_num, 1, head_size]
    BaseWeight<T> *qkv, // bias, [qkv_head_num * head_size]
    TensorWrapper<int> *layer_id, // [layer_num]
    TensorWrapper<T> *k_cache, // [layer_num, batch_size, kv_num_heads, max_seq_len, head_size]
    TensorWrapper<T> *v_cache, // [layer_num, batch_size, kv_num_heads, max_seq_len, head_size]
    TensorWrapper<bool> *finished, // [batch_size]
    TensorWrapper<int> *step, // ?[max_seq_len]
    TensorWrapper<T> *mha_output, // [batch_size, num_heads, head_size]
    LlamaAttentionStaticParams *static_params
) {
    const int batch_size = qkv_buf->shape[0];
    const int qkv_head_num = qkv_buf->shape[1];
    const int head_size = qkv_buf->shape[2];

    const int kv_head_num = k_cache->shape[2];
    const int max_seq_len = k_cache->shape[3];
    const int head_num = qkv_head_num - 2 * kv_head_num;

    const int cur_step = step->getVal();
    const int layer = layer_id->getVal();
    const int layer_offset = layer * max_seq_len * batch_size * kv_head_num * head_size;

    const int smem_size_bytes = head_size * sizeof(T) + cur_step * sizeof(float);
    T *const qkv_data = qkv_buf->data;
    T *const q = qkv_data;
    T *const k = qkv_data + head_num * head_size;
    T *const v = qkv_data + (head_num + kv_head_num) * head_size;

    const int rotary_embedding_dim = static_params->rotary_embedding_dim;
    const float rotary_embedding_base = static_params->rotary_embedding_base;

    dim3 grid(head_num * batch_size);
    dim3 block(head_size);

    maskedMultiHeadAttention<T><<<grid, block, smem_size_bytes>>>(
        q, 
        k, 
        v, 
        qkv->bias, 
        k_cache->data + layer_offset,
        v_cache->data + layer_offset, 
        mha_output->data, 
        batch_size,
        head_num, 
        kv_head_num, 
        max_seq_len, 
        head_size, 
        cur_step,
        rotary_embedding_dim,
        rotary_embedding_base
    );
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(mha_output->data, true);
#else
#endif
}

template void launchDecoderMaskedMultiHeadAttention(
    TensorWrapper<float> *qkv_buf,
    BaseWeight<float> *qkv,
    TensorWrapper<int> *layer_id,
    TensorWrapper<float> *k_cache,
    TensorWrapper<float> *v_cache,
    TensorWrapper<bool> *finished,
    TensorWrapper<int> *step,
    TensorWrapper<float> *mha_output,
    LlamaAttentionStaticParams *static_params
);

template void launchDecoderMaskedMultiHeadAttention(
    TensorWrapper<half> *qkv_buf,
    BaseWeight<half> *qkv,
    TensorWrapper<int> *layer_id,
    TensorWrapper<half> *k_cache,
    TensorWrapper<half> *v_cache,
    TensorWrapper<bool> *finished,
    TensorWrapper<int> *step,
    TensorWrapper<half> *mha_output,
    LlamaAttentionStaticParams *static_params
);