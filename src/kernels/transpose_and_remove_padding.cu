#include "hip/hip_runtime.h"
#include <iostream>
#include "../utils/cuda_debug_utils.cuh"
#include "includes/transpose_and_remove_padding.h"

/* 
    [bs, head nums, seqlen, head size] => 
    [bs, seqlen, head nums, head size] => 
    [num tokens, head nums, head size]
    padding_offset.shape = [num_tokens]

    dim3 grid(num_tokens);
    dim3 block(std::min(head_num * head_size, 1024));
*/

template <typename T>
__global__ void fusedTransposeAndRemovePadding(
    const T *src,                // [bs, head nums, seqlen, head size]
    T *dst,                      // [num tokens, head nums, head size]
    const int num_tokens,
    const int batch_size,
    const int seq_len,
    const int head_num,
    const int head_size,
    const int *padding_offset    // [num_tokens]
) {
    const int token_id = blockIdx.x;
    // Map to input id
    const int batch_id = (blockIdx.x + padding_offset[token_id]) / seq_len;
    const int seq_id = (blockIdx.x + padding_offset[token_id]) % seq_len;
    const int tid = threadIdx.x;
    const int hidden_units = head_num * head_size;

    // Compute the offset of transpose and remove padding before or after
    const int src_base_offset = batch_id * head_num * seq_len * head_size + seq_id * head_size;
    const int dst_base_offset = token_id * head_num * head_size;

    #pragma unroll
    for (int i = tid; i < hidden_units; i += blockDim.x) {
        const int head_id = i / head_size;
        const int head_inner_id = i % head_size;
        dst[dst_base_offset + i] = src[src_base_offset + head_id * seq_len * head_size + head_inner_id];
    }
}

template <typename T>
void launchFusedTransposeAndRemovePadding(
    TensorWrapper<T> *padded_qkv_buf,  // [bs, head nums, seqlen, head size]
    TensorWrapper<int> *padding_offset,      // [num_tokens]
    TensorWrapper<T> *lineared_qkv_buf // [num tokens, head nums, head size]
) {
    const int batch_size = padded_qkv_buf->shape[0];
    const int head_num = padded_qkv_buf->shape[1];
    const int seq_len = padded_qkv_buf->shape[2];
    const int head_size = padded_qkv_buf->shape[3];
    const int num_tokens = lineared_qkv_buf->shape[0];

    dim3 grid(num_tokens);
    dim3 block(std::min(head_num * head_size, 1024));

    fusedTransposeAndRemovePadding<T><<<grid, block>>>(
        padded_qkv_buf->data,
        lineared_qkv_buf->data,
        num_tokens,
        batch_size,
        seq_len,
        head_num,
        head_size,
        padding_offset->data
    );

#ifdef PRINT_DATA
    print_data<<<1, 1>>>(lineared_qkv_buf->data);
#endif
}

template void launchFusedTransposeAndRemovePadding(
    TensorWrapper<float> *padded_qkv_buf,
    TensorWrapper<int> *padding_offset,
    TensorWrapper<float> *lineared_qkv_buf
);

template void launchFusedTransposeAndRemovePadding(
    TensorWrapper<half> *padded_qkv_buf,
    TensorWrapper<int> *padding_offset,
    TensorWrapper<half> *lineared_qkv_buf
);
