#include "hip/hip_runtime.h"
#include "src/kernels/includes/attn_softmax.h"
#include "src/utils/tensor.h"
// #include "src/utils/cuda_debug_utils.cuh"
#include <float.h>
#include <math.h>
#include <assert.h>
#include <stdio.h>

template <typename T>
struct SumOp {
    __device__ __forceinline__ T operator()(const T &a, const T &b) const { 
        return a + b;
    }
};

template <typename T>
struct MaxOp {
    __device__ __forceinline__ T operator()(const T &a, const T &b) const { 
        return max(a, b); 
    }
};

template <template <typename> class ReductionOp, typename T>
__device__ __forceinline__ T warpReduce(T val) {
    #pragma unroll
    for (int mask = 16; mask > 0; mask /= 2) {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}

template <template <typename> class ReductionOp, typename T>
__device__ T blockReduce(T val) {
    int tid = threadIdx.x;
    int warp_id = tid >> 5;
    int lane_id = tid & 31;
    int warp_nums = (blockDim.x + 31) >> 5;
    static __shared__ T warp[32]; // threads in a block must less 1024
    val = warpReduce<ReductionOp, T>(val);
    if (lane_id == 0) {
        warp[warp_id] = val;
    }
    __syncthreads();
    T warp_val = tid < warp_nums ? warp[tid] : 0;
    return warpReduce<ReductionOp, T>(warp_val);
}

template <typename T, int NUMS_PER_THREAD_PER_ROW>
__global__ void scaleMaskAndSoftmax_float(T *attn_score,
                                          T *qk,
                                          T *mask,
                                          int batch_size,
                                          int head_nums,
                                          int q_len,
                                          int k_len,
                                          float scale) {
    int batch_id = blockIdx.y;
    int head_id = blockIdx.z;
    if (threadIdx.x >= k_len) {
        return;
    }

    __shared__ float inv_sum, s_max;
    for (int row_start = blockIdx.x; row_start < q_len; row_start += gridDim.x) {
        int qk_offset = 0;
        int mask_offset = 0;
        T qk_data = static_cast<T>(0);
        T mask_data = static_cast<T>(0);
        T thread_max = FLT_MIN;
        T data[NUMS_PER_THREAD_PER_ROW];

        for (int col_start = 0; col_start < NUMS_PER_THREAD_PER_ROW; ++col_start) {
            qk_offset = batch_id * head_nums * q_len * k_len + head_id * q_len * k_len + row_start * k_len + col_start * blockDim.x + threadIdx.x;
            qk_data = qk[qk_offset];

            mask_offset = batch_id * q_len * k_len + row_start * k_len + col_start * blockDim.x + threadIdx.x;
            mask_data = mask[mask_offset];

            data[col_start] = scale * qk_data + (1 - mask_data) * (-10000.0f);
            thread_max = fmax(data[col_start], thread_max);
        }

        T max_val = blockReduce<MaxOp, T>(thread_max);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        T thread_sum = 0.0f;
        for (int col_start = 0; col_start < NUMS_PER_THREAD_PER_ROW; ++col_start) {
            qk_offset = batch_id * head_nums * q_len * k_len + head_id * q_len * k_len + row_start * k_len + col_start * blockDim.x + threadIdx.x;
            mask_offset = batch_id * q_len * k_len + row_start * k_len + col_start * blockDim.x + threadIdx.x;
            data[col_start] = expf(data[col_start] - s_max);
            thread_sum += data[col_start];
        }

        T sum = blockReduce<SumOp, T>(thread_sum);
        if (threadIdx.x == 0) {
            inv_sum = 1 / (sum + 1e-6f);
        }
        __syncthreads();

        for (int col_start = 0; col_start < NUMS_PER_THREAD_PER_ROW; ++col_start) {
            qk_offset = batch_id * head_nums * q_len * k_len + head_id * q_len * k_len + row_start * k_len + col_start * blockDim.x + threadIdx.x;
            attn_score[qk_offset] = (data[col_start] * inv_sum);
        }
    }
}

template <typename T_half, int NUMS_PER_THREAD_PER_ROW>
__global__ void scaleMaskAndSoftmax_half(T_half *attn_score,
                                         T_half *qk,
                                         T_half *mask,
                                         int batch_size,
                                         int head_nums,
                                         int q_len,
                                         int k_len,
                                         float scale) {
    int batch_id = blockIdx.y;
    int head_id = blockIdx.z;
    int vec_size = Vec<T_half>::size;
    using Vec_t = typename Vec<T_half>::Type;

    Vec_t *attn_score_vec = reinterpret_cast<Vec_t *>(attn_score);
    Vec_t *qk_buf_vec = reinterpret_cast<Vec_t *>(qk);
    Vec_t *attn_mask_vec = reinterpret_cast<Vec_t *>(mask);
    Vec_t ONE = scalar_cast_vec<Vec_t>(__float2half(1.0f));
    Vec_t NEG_INF = scalar_cast_vec<Vec_t>(__float2half(-10000.0f));
    Vec_t scale_vec = scalar_cast_vec<Vec_t>(__float2half(scale));

    __shared__ float inv_sum, s_max;
    if (threadIdx.x * vec_size >= k_len) {
        return;
    }

    for (int row_start = blockIdx.x; row_start < q_len; row_start += gridDim.x) {
        int qk_offset = 0;
        int mask_offset = 0;
        Vec_t qk_data;
        Vec_t mask_data;
        float thread_max = FLT_MIN;
        Vec_t data[NUMS_PER_THREAD_PER_ROW];

        for (int col_start = 0; col_start < NUMS_PER_THREAD_PER_ROW; ++col_start) {
            qk_offset = batch_id * head_nums * q_len * k_len / 2 + head_id * q_len * k_len / 2 + row_start * k_len / 2 + col_start * blockDim.x + threadIdx.x;
            qk_data = qk_buf_vec[qk_offset];

            mask_offset = batch_id * q_len * k_len / 2 + row_start * k_len / 2 + col_start * blockDim.x + threadIdx.x;
            mask_data = attn_mask_vec[mask_offset];
            Vec_t mask_vec_reg = __hmul2(__hsub2(ONE, mask_data), NEG_INF);

            data[col_start] = __hadd2(__hmul2(scale_vec, qk_data), mask_vec_reg);
            thread_max = fmax(fmax((float)data[col_start].x, (float)data[col_start].y), thread_max);
        }

        float max_val = blockReduce<MaxOp, float>(thread_max);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        float thread_sum = 0.0f;
        for (int col_start = 0; col_start < NUMS_PER_THREAD_PER_ROW; ++col_start) {
            data[col_start] = h2exp(__hsub2(data[col_start], scalar_cast_vec<Vec_t>(s_max)));
            thread_sum += (float)(__hadd(data[col_start].x, data[col_start].y));
        }

        float sum = blockReduce<SumOp, float>(thread_sum);
        if (threadIdx.x == 0) {
            inv_sum = 1 / (sum + 1e-6f);
        }
        __syncthreads();

        for (int col_start = 0; col_start < NUMS_PER_THREAD_PER_ROW; ++col_start) {
            qk_offset = batch_id * head_nums * q_len * k_len / 2 + head_id * q_len * k_len / 2 + row_start * k_len / 2 + col_start * blockDim.x + threadIdx.x;
            attn_score_vec[qk_offset] = __hmul2(data[col_start], scalar_cast_vec<Vec_t>(inv_sum));
        }
    }
}

template <typename T>
void launchScaleMaskAndSoftmax(TensorWrapper<T> *qk,
                               TensorWrapper<T> *mask,
                               TensorWrapper<T> *attn_score,
                               float scale) {
    // attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
    // qk,                 (batch_size, head_num, q_length, k_length), QK^T.
    // attention_mask,     (batch_size, q_length, k_length), attention mask.
    int q_length = qk->shape[2];
    int batch_size = qk->shape[0];
    int head_nums = qk->shape[1];
    int k_length = qk->shape[3];
    bool is_half = sizeof(T) == 2;
    // TODO: should enhance it by padding to support odd ones
    if (is_half) {
        LLM_CHECK_WITH_INFO(k_length % 2 == 0, "Currently, K_len should be divided by 2 under half type!");
    }
    dim3 grid(q_length, batch_size, head_nums);
    dim3 block((k_length + 32 - 1) / 32 * 32); // align with 32x threads

    if (is_half) {
        if (block.x > 2048 && block.x <= 4096) {
            constexpr int NUMS_PER_THREAD_PER_ROW = 4;
            block.x /= 4 * 2;
            block.x = (block.x + 32 - 1) / 32 * 32;
            assert(block.x < 1024);
            scaleMaskAndSoftmax_half<half, NUMS_PER_THREAD_PER_ROW><<<grid, block>>>(
                reinterpret_cast<half *>(attn_score->data),
                reinterpret_cast<half *>(qk->data),
                reinterpret_cast<half *>(mask->data),
                batch_size,
                head_nums,
                q_length,
                k_length,
                scale
            );
        } else if (block.x > 1024) {
            constexpr int NUMS_PER_THREAD_PER_ROW = 2;
            block.x /= 2 * 2;
            block.x = (block.x + 32 - 1) / 32 * 32;
            assert(block.x < 1024);
            scaleMaskAndSoftmax_half<half, NUMS_PER_THREAD_PER_ROW><<<grid, block>>>(
                reinterpret_cast<half *>(attn_score->data),
                reinterpret_cast<half *>(qk->data),
                reinterpret_cast<half *>(mask->data),
                batch_size,
                head_nums,
                q_length,
                k_length,
                scale
            );
        } else {
            constexpr int NUMS_PER_THREAD_PER_ROW = 1;
            block.x /= 2;
            assert(block.x < 1024);
            scaleMaskAndSoftmax_half<half, NUMS_PER_THREAD_PER_ROW><<<grid, block>>>(
                reinterpret_cast<half *>(attn_score->data),
                reinterpret_cast<half *>(qk->data),
                reinterpret_cast<half *>(mask->data),
                batch_size,
                head_nums,
                q_length,
                k_length,
                scale
            );
        }
    } else {
        if (block.x > 2048 && block.x <= 4096) {
            constexpr int NUMS_PER_THREAD_PER_ROW = 4;
            block.x /= 4;
            block.x = (block.x + 32 - 1) / 32 * 32;
            assert(block.x < 1024);
            scaleMaskAndSoftmax_float<float, NUMS_PER_THREAD_PER_ROW><<<grid, block>>>(
                reinterpret_cast<float *>(attn_score->data),
                reinterpret_cast<float *>(qk->data),
                reinterpret_cast<float *>(mask->data),
                batch_size,
                head_nums,
                q_length,
                k_length,
                scale
            );
        } else if (block.x > 1024) {
            constexpr int NUMS_PER_THREAD_PER_ROW = 2;
            block.x /= 2;
            block.x = (block.x + 32 - 1) / 32 * 32;
            assert(block.x < 1024);
            scaleMaskAndSoftmax_float<float, NUMS_PER_THREAD_PER_ROW><<<grid, block>>>(
                reinterpret_cast<float *>(attn_score->data),
                reinterpret_cast<float *>(qk->data),
                reinterpret_cast<float *>(mask->data),
                batch_size,
                head_nums,
                q_length,
                k_length,
                scale
            );
        } else {
            constexpr int NUMS_PER_THREAD_PER_ROW = 1;
            block.x /= 1;
            assert(block.x < 1024);
            scaleMaskAndSoftmax_float<float, NUMS_PER_THREAD_PER_ROW><<<grid, block>>>(
                reinterpret_cast<float *>(attn_score->data),
                reinterpret_cast<float *>(qk->data),
                reinterpret_cast<float *>(mask->data),
                batch_size,
                head_nums,
                q_length,
                k_length,
                scale
            );
        }
    }
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(attn_score->data);
#endif
}

template void launchScaleMaskAndSoftmax(TensorWrapper<float> *qk,
                                        TensorWrapper<float> *mask,
                                        TensorWrapper<float> *attn_score,
                                        float scale);

template void launchScaleMaskAndSoftmax(TensorWrapper<half> *qk,
                                        TensorWrapper<half> *mask,
                                        TensorWrapper<half> *attn_score,
                                        float scale);
