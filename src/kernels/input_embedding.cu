#include "hip/hip_runtime.h"
#include <stdio.h>
#include "src/kernels/includes/input_embedding.h"

template<typename T>
__global__ void embeddingFunctor(const int *input_ids, T *output, 
                                 const T *embed_table,
                                 const int max_context_token_num,
                                 const int hidden_size) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = gid; i < max_context_token_num * hidden_size; i += stride) {
        int token_val = input_ids[i / hidden_size];
        int feature_vector_idx = i % hidden_size;
        output[i] = *(embed_table + token_val * hidden_size + feature_vector_idx);
    }
}

template<typename T>
void launchInputEmbedding(TensorWrapper<int> *input_ids,    // INT [token num]
                          TensorWrapper<T> *output,       // FP32 [token num, hidden_size] = [token num, 4096]
                          EmbeddingWeight<T> *embed_table) { // FP32 [vocal_size, hidden_size]
    const int blockSize = 256;
    const int max_context_token_num = output->shape[0]; // token num
    const int hidden_size = output->shape[1];
    const int gridSize = 2048;
    
    LLM_CHECK_WITH_INFO(max_context_token_num == input_ids->shape[0], 
                        "input ids 1st shape should equal to 1st shape of output");
    
    embeddingFunctor<T><<<gridSize, blockSize>>>(input_ids->data,
                                                 output->data,
                                                 embed_table->data,
                                                 max_context_token_num,
                                                 hidden_size);
                                                 
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(output->data);
#endif
}

// Explicit template instantiation
template void launchInputEmbedding(TensorWrapper<int>* input_ids,    
                                   TensorWrapper<float>* output,       
                                   EmbeddingWeight<float>* embed_table);
                                   
template void launchInputEmbedding(TensorWrapper<int>* input_ids,    
                                   TensorWrapper<half>* output,
                                   EmbeddingWeight<half>* embed_table);
