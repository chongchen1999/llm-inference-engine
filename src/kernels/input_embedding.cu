#include "hip/hip_runtime.h"
#include <stdio.h>
#include "src/kernels/includes/input_embedding.h"

template<typename T>
__global__ void embeddingFunctor(const int *input_ids, T *output, 
                                 const T *embed_table,
                                 const int max_context_token_num,
                                 const int hidden_size) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    #pragma unroll
    for (int i = gid; i < max_context_token_num * hidden_size; i += stride) {
        int token_val = input_ids[i / hidden_size];
        int feature_vector_idx = i % hidden_size;
        output[i] = embed_table[token_val * hidden_size + feature_vector_idx];
    }
}

template<typename T>
void launchInputEmbedding(TensorWrapper<int> *input_ids,    // INT [token num]
                          TensorWrapper<T> *output,       // T [token num, hidden_size] = [token num, 4096]
                          EmbeddingWeight<T> *embed_table) { // T [vocab_size, hidden_size]
    const int block_size = 256;
    const int max_context_token_num = output->shape[0]; // token num
    const int hidden_size = output->shape[1];
    const int grid_size = 2048;
    
    LLM_CHECK_WITH_INFO(max_context_token_num == input_ids->shape[0], 
                        "Input IDs 1st shape should equal 1st shape of output");
    
    embeddingFunctor<T><<<grid_size, block_size>>>(input_ids->data,
                                                   output->data,
                                                   embed_table->data,
                                                   max_context_token_num,
                                                   hidden_size);
                                                 
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(output->data);
#endif
}

// Explicit template instantiation
template void launchInputEmbedding(TensorWrapper<int> *input_ids,    
                                   TensorWrapper<float> *output,       
                                   EmbeddingWeight<float> *embed_table);
                                   
template void launchInputEmbedding(TensorWrapper<int> *input_ids,    
                                   TensorWrapper<half> *output,
                                   EmbeddingWeight<half> *embed_table);
